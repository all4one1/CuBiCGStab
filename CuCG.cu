#include "hip/hip_runtime.h"
#pragma once
#include "CuCG.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__device__ double rs_old = 1, rs_new = 1, alpha = 1, beta = 1, buffer = 1, buffer2 = 1, omega = 1;


#define KERNEL(func) func<<< blocks, threads>>>

BiCGSTAB::BiCGSTAB() {};
BiCGSTAB::BiCGSTAB(unsigned int N_, double* x, double* x0, double* b,
	SparseMatrixCuda& A, CudaLaunchSetup kernel_setting, unsigned int reduction_threads)
{
	N = N_;
	Nbytes = N * sizeof(double);
	threads = kernel_setting.Block1D.x;
	blocks = kernel_setting.Grid1D.x;

	#define alloc_(ptr) hipMalloc((void**)&##ptr, Nbytes);  hipMemset(ptr, 0, Nbytes); 
	alloc_(r); alloc_(r_hat); alloc_(p); alloc_(t); alloc_(s); alloc_(v);

	CR = new CudaReductionM(N, reduction_threads);
	make_graph(x, x0, b, A);
}
void BiCGSTAB::solve_directly(double* x, double* x0, double* b, SparseMatrixCuda& A)
{
	double rs_host = 1;  k = 0;
	hipMemset(x, 0, Nbytes);

	// r = b - Ax
	KERNEL(vector_minus_matrix_dot_vector)(r, b, A, x, N);
	// r_hat = r
	KERNEL(vector_set_to_vector)(r_hat, r, N);
	// p = r
	KERNEL(vector_set_to_vector)(p, r, N);

	// rs = r_hat * r
	CR->reduce(r_hat, r, true, ExtraAction::compute_rs_old);

	auto single_iteration = [&]()
	{

		// rs_new = r_hat * r; 		// beta =  (rs_new / rs_old) * (alpha / omega)		// rs_old = rs_new
		CR->reduce(r_hat, r, false, ExtraAction::compute_rs_new_and_beta);

		// p = r + beta * ( p - omega * v)
		KERNEL(vector_add_2vectors)(p, r, p, v, N, KernelCoefficient::beta_and_omega);

		// v = Ap
		KERNEL(matrix_dot_vector)(v, A, p, N);

		// alpha = rs_new / (r_hat * v)
		CR->reduce(r_hat, v, false, ExtraAction::compute_alpha);

		// s = r - alpha * v
		KERNEL(vector_minus_vector)(s, r, v, N, KernelCoefficient::alpha);

		// t = A * s
		KERNEL(matrix_dot_vector)(t, A, s, N);

		// omega = (t * s) / (t * t)

		CR->reduce(t, s, false, ExtraAction::compute_buffer);
		CR->reduce(t, t, false, ExtraAction::compute_omega);

		// x = x + alpha * p + omega * s
		KERNEL(vector_add_2vectors)(x, x, p, s, N, KernelCoefficient::alpha_and_omega);

		// r = s - omega * t
		KERNEL(vector_minus_vector)(r, s, t, N, KernelCoefficient::omega);
	};


	while (true)
	{
		k++;	if (k > 1000000) break;

		single_iteration();

		// check exit by r^2
		if (k < 20 || k % 50 == 0)
		{
			rs_host = CR->reduce(r, r, true, ExtraAction::NONE);
			//if (k > 100000) break;
			if (abs(rs_host) < eps) break;
		}

		//if (k == 20000) break;
		if (k % 1000 == 0) std::cout << k << " " << abs(rs_host) << std::endl;
	}

	//std::cout << k << " " << abs(rs_host) << std::endl;
}
void BiCGSTAB::make_graph(double* x, double* x0, double* b, SparseMatrixCuda& A)
{
	KernelCoefficient action;

	// 1. rs_new = r_hat * r; 		// beta =  (rs_new / rs_old) * (alpha / omega)		// rs_old = rs_new
	graph.add_graph_as_node(CR->make_graph(r_hat, r, false, ExtraAction::compute_rs_new_and_beta));

	// 2. p = r + beta * ( p - omega * v)
	{
		action = KernelCoefficient::beta_and_omega;
		void* args[] = { &p, &r, &p, &v, &N, &action };
		graph.add_kernel_node(threads, blocks, vector_add_2vectors, args);
	}

	// 3. v = Ap
	{
		void* args[] = { &v, &A, &p, &N };
		graph.add_kernel_node(threads, blocks, matrix_dot_vector, args);
	}

	// 4. alpha = rs_new / (r_hat * v)
	graph.add_graph_as_node(CR->make_graph(r_hat, v, false, ExtraAction::compute_alpha));

	// 5. s = r - alpha * v
	{
		action = KernelCoefficient::alpha;
		void* args[] = { &s, &r, &v, &N, &action };
		graph.add_kernel_node(threads, blocks, vector_minus_vector, args);
	}

	// 6. t = A * s
	{
		void* args[] = { &t, &A, &s, &N };
		graph.add_kernel_node(threads, blocks, matrix_dot_vector, args);
	}

	// 7. omega = (t * s) / (t * t)
	graph.add_graph_as_node(CR->make_graph(t, s, false, ExtraAction::compute_buffer));
	graph.add_graph_as_node(CR->make_graph(t, t, false, ExtraAction::compute_omega));

	// 8. x = x + alpha * p + omega * s
	{
		action = KernelCoefficient::alpha_and_omega;
		void* args[] = { &x, &x, &p, &s, &N, &action };
		graph.add_kernel_node(threads, blocks, vector_add_2vectors, args);
	}

	// 9. r = s - omega * t
	{
		action = KernelCoefficient::omega;
		void* args[] = { &r, &s, &t, &N, &action };
		graph.add_kernel_node(threads, blocks, vector_minus_vector, args);
	}

	graph.instantiate();
}
void BiCGSTAB::solve_with_graph(double* x, double* x0, double* b, SparseMatrixCuda& A)
{
	double rs_host = 1;  k = 0;
	hipMemset(x, 0, Nbytes);

	// r = b - Ax
	KERNEL(vector_minus_matrix_dot_vector)(r, b, A, x, N);
	// r_hat = r
	KERNEL(vector_set_to_vector)(r_hat, r, N);
	// p = r
	KERNEL(vector_set_to_vector)(p, r, N);
	// rs = r_hat * r
	CR->reduce(r_hat, r, false, ExtraAction::compute_rs_old);

	while (true)
	{
		k++;	if (k > 1000000) break;
		graph.launch();

		// check exit by r^2
		if (k < 20 || k % 50 == 0)
		{
			rs_host = CR->reduce(r, r, true, ExtraAction::NONE);
			//if (k > 100000) break;
			if (abs(rs_host) < eps) break;
		}

		//if (k == 20000) break;
		if (k % 1000 == 0) std::cout << k << " " << abs(rs_host) << std::endl;
	}
	std::cout << k << " " << abs(rs_host) << std::endl;
}